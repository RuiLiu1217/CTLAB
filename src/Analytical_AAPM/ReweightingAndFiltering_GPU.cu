#include "hip/hip_runtime.h"
/**
 * Wake Forest University Health Science
 * University of Massachusetts Lowell
 *
 * Organization:
 * 	Wake Forest University
 *
 * 	reWeiAdFiltr.cpp
 * 	Matlab mex routine for the GPU based reweighting
 * 	and filtering for analytical helical CT
 * 	reconstruction
 *
 * 	Author: Rui Liu
 * 	Email: liurui1217@gmail.com
 * 	Date: 2016-08-04
 *
 * 	Version 1.0
 */

#include "mex.h"
#include "matrix.h"

#include <iostream>

#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <hipfft/hipfft.h>
// includes, project
#include <hip/hip_runtime.h>

#include <iostream>
typedef float2 Complex;


struct genKernel
{
	float _dYL;
	genKernel(const float _dyl):_dYL(_dyl){}

	__host__ __device__ float operator()(float n) const
	{
		return -2.0 / (9.869604401089358 * (4.0 * n * n - 1.0)) / _dYL;
	}
};

// Generate the Hilbert Filtering Kernel

__global__ void copyKernel(hipfftComplex* output,const float* HS, const int NN)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < NN)
	{
		output[id].x = HS[id];
		output[id].y = 0;
	}
}

__global__ void copyKernel(Complex* output, const hipfftComplex* HS, const int NN)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < NN)
	{
		output[id].x = HS[id].x;
		output[id].y = HS[id].y;
	}
}
__global__ void copyKernel(float* output, const hipfftComplex* HS, const int NN)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < NN)
	{
		output[id] = HS[id].x;

	}
}
int genHilbertKer(
		thrust::device_vector<hipfftComplex>& FtS,
		const int YL,
		const float dYL)
{
	thrust::device_vector<float> n(2 * YL + 1, 0);
	thrust::sequence(n.begin(),n.end(),static_cast<float>(-YL));
	thrust::transform(n.begin(),n.end(),n.begin(),genKernel(dYL));
	const int NN = pow(2.0,ceil(log2(YL * 3.0)));

	FtS.resize(NN);
	thrust::device_vector<float> HS(NN,0);
	thrust::copy(n.begin(),n.begin() + YL,HS.end() - YL);
	thrust::copy(n.begin()+YL,n.end(),HS.begin());

	// To Do the CUFFT
	hipfftHandle plan;
	dim3 blk(1024);
	dim3 gid((NN + blk.x - 1) / blk.x);
	copyKernel<<<gid,blk>>>(thrust::raw_pointer_cast(&FtS[0]),thrust::raw_pointer_cast(&HS[0]), NN);
	hipfftPlan1d(&plan, NN, HIPFFT_C2C,1);
	hipfftExecC2C(plan, thrust::raw_pointer_cast(&FtS[0]), thrust::raw_pointer_cast(&FtS[0]), HIPFFT_FORWARD);
	hipfftDestroy(plan);
	HS.clear();
	n.clear();
	return NN;

}

__global__ void copyExpandedProjectionData(hipfftComplex* output, const float* input,
		const int YL, const int ZL, const int ViewN, const int NN)
{
	int curPos = threadIdx.x + blockIdx.x * blockDim.x;
	int curBatch = threadIdx.y + blockIdx.y * blockDim.y;
	if(curPos < YL && curBatch < ZL * ViewN)
	{
		output[curBatch * NN + curPos].x = input[curBatch * YL + curPos];
		output[curBatch * NN + curPos].y = 0;
	}

}

__global__ void multiplyProjectionWithKernel(hipfftComplex* proj, const hipfftComplex* kernel, const int kernelLength, const int batchSize)
{
	int kerIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int batIdx = threadIdx.y + blockIdx.y * blockDim.y;
	//__shared__ float kk[KSIZE];
	//kk[threadIdx.x] = kernel[kerIdx];
	//__syncthreads();
	if(kerIdx < kernelLength && batIdx < batchSize)
	{
		hipfftComplex kk = kernel[kerIdx];
		hipfftComplex res;
		res.x = proj[batIdx * kernelLength + kerIdx].x * kk.x - proj[batIdx * kernelLength + kerIdx].y * kk.y;
		res.y = proj[batIdx * kernelLength + kerIdx].x * kk.y + proj[batIdx * kernelLength + kerIdx].y * kk.x;
		proj[batIdx * kernelLength + kerIdx] = res;
		//proj[batIdx * kernelLength + kerIdx].y *= kk;
	}
}

__global__ void cutProjectionData(float* fpwd, hipfftComplex* proj, const int YL, const int NN, const int batSize)
{
	int curIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int batIdx = threadIdx.y + blockIdx.y * blockDim.y;
	if(curIdx < YL && batIdx < batSize)
	{
		fpwd[batIdx * YL + curIdx] = proj[batIdx * NN + curIdx].x / NN;
	}
}

// Let the projection data stored in the addressing order:
// 1. detector cell transversal direction (YL)
// 2. vertical direction (ZL)
// 3. view index (ViewN)
void filtering(
		thrust::device_vector<float>& fpwd, // Filtered projection data
		const thrust::device_vector<float>& Proj, // Projection data
		const int YL, const int ZL, const int ViewN, // Size of the projection data
		const float dYL)
{
	thrust::device_vector<hipfftComplex> FtS;
	int NN = genHilbertKer(FtS, YL, dYL);
	//Expand the projection data
	thrust::device_vector<hipfftComplex> exProj(NN * ZL * ViewN);
	dim3 copyExpBlk(32,32);
	dim3 copyExpGid(
			(YL + copyExpBlk.x - 1) / copyExpBlk.x,
			(ZL * ViewN + copyExpBlk.y - 1) / copyExpBlk.y);
	copyExpandedProjectionData<<<copyExpGid, copyExpBlk>>>(
			thrust::raw_pointer_cast(&exProj[0]),
			thrust::raw_pointer_cast(&Proj[0]),
			YL, ZL, ViewN, NN);
	// Forward Batch FFT
	hipfftHandle plan;
	hipfftPlan1d(&plan, NN, HIPFFT_C2C, ZL * ViewN);
	hipfftExecC2C(plan, thrust::raw_pointer_cast(&exProj[0]),
			thrust::raw_pointer_cast(&exProj[0]),HIPFFT_FORWARD);
	// Multiply with the kernel
	dim3 multBlk(32,32);
	dim3 multGid(
			(NN + multBlk.x - 1) / multBlk.x,
			(ZL * ViewN + multBlk.y - 1) / multBlk.y);

	multiplyProjectionWithKernel<<<multGid,multBlk>>>(thrust::raw_pointer_cast(&exProj[0]),
			thrust::raw_pointer_cast(&FtS[0]), NN, ZL * ViewN);

	// Back batch FFT
	hipfftExecC2C(plan, thrust::raw_pointer_cast(&exProj[0]),
				thrust::raw_pointer_cast(&exProj[0]),HIPFFT_BACKWARD);

	// Cut the data
	cutProjectionData<<<copyExpGid, copyExpBlk>>>(thrust::raw_pointer_cast(&fpwd[0]),
			thrust::raw_pointer_cast(&exProj[0]),
			YL, NN, ZL * ViewN);

	hipfftDestroy(plan);
	FtS.clear();
	exProj.clear();
}

__global__ void preWeighting_ker(float* Proj,
		const int YL,
		const int ZL,
		const int ViewN,
		const float PLC,
		const float ZLC,
		const float dYL,
		const float dZL,
		const float SO)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int v = threadIdx.z + blockIdx.z * blockDim.z;
	if(j < YL && k < ZL && v < ViewN)
	{
		const float t = (j - PLC) * dYL;
		const float b = (k - ZLC) * dZL;
		const float wei = SO * SO / sqrtf(SO * SO * (SO * SO + b * b) - b * t * b * t);
		Proj[(v * ZL + k) * YL + j] *= wei;
	}
}

void preWeighting(
		thrust::device_vector<float>& Proj,
		const int YL,
		const int ZL,
		const int ViewN,
		const float PLC,
		const float ZLC,
		const float dYL,
		const float dZL,
		const float SO)
{
	dim3 blk(16,4,4);
	dim3 gid(
			(YL + blk.x - 1) / blk.x,
			(ZL + blk.y - 1) / blk.y,
			(ViewN + blk.z - 1) / blk.z);
	preWeighting_ker<<<gid,blk>>>(
		thrust::raw_pointer_cast(&Proj[0]),
		YL, ZL, ViewN, PLC, ZLC, dYL, dZL, SO);

}



//////////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void addressOrder(
		T* proj_ZYV,
		const T* proj_YZV,
		const int YL, const int ZL, const int ViewN)
{
	int zIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int yIdx = threadIdx.y + blockIdx.y * blockDim.y;
	int vIdx = threadIdx.z + blockIdx.z * blockDim.z;
	if(zIdx < ZL && yIdx < YL && vIdx < ViewN)
	{
		proj_ZYV[(vIdx * YL + yIdx) * ZL + zIdx] =
				proj_YZV[(vIdx * ZL + zIdx) * YL + yIdx];

	}
}

template<typename T>
__global__ void addressOrder_2(
		T* proj_YZV,
		const T* proj_ZYV,
		const int YL, const int ZL, const int ViewN)
{
	int yIdx = threadIdx.x + blockIdx.x * blockDim.x;
	int zIdx = threadIdx.y + blockIdx.y * blockDim.y;
	int vIdx = threadIdx.z + blockIdx.z * blockDim.z;
	if(zIdx < ZL && yIdx < YL && vIdx < ViewN)
	{
		proj_YZV[(vIdx * ZL + zIdx) * YL + yIdx]
		         = proj_ZYV[(vIdx * YL + yIdx) * ZL + zIdx];

	}
}



extern "C"
void filtering(float* hfpwd,
		const float* hProj,
		const int YL, const int ZL, const int ViewN,
		const float PLC, const float ZLC,
		const float dYL, const float dZL,
		const float SO, const int GPUID)
{
    hipSetDevice(GPUID);
	thrust::device_vector<float> Proj(hProj, hProj + YL * ZL * ViewN);
	preWeighting(Proj, YL, ZL, ViewN, PLC,
			 ZLC, dYL, dZL, SO);
	thrust::device_vector<float> fpwd(YL * ZL * ViewN, 0);
	filtering(fpwd,Proj, YL, ZL, ViewN, dYL);
	thrust::copy(fpwd.begin(),fpwd.end(),hfpwd);
	
}


// The function routine should be called
// FPWD = reWeihAdFiltr(Proj, PLC, ZLC, dYL, dZL, SO);
void mexFunction(
		int nlhs, mxArray* plhs[],
		int nrhs, const mxArray* prhs[])
{
	// YL, ZL, ViewN should be calculated from Proj
	if(nlhs != 1 || nrhs != 6)
	{
		std::cerr<<"Wrong number of parameters\n";
		std::cerr<<"This function requires 6 input parameter and provides one return value\n";
		exit(-1);
	}
	const mwSize* siz = mxGetDimensions(prhs[0]);
	const int YL = siz[0];
	const int ZL = siz[1];
	const int ViewN = siz[2];
	// Generate a new array
	plhs[0] = mxCreateNumericArray(3,siz,mxSINGLE_CLASS,mxREAL);
	float* fpwd = (float*)mxGetPr(plhs[0]);
	const float* Proj = (float*)mxGetPr(prhs[0]);
	const float PLC = *((float*)mxGetData(prhs[1]));
	const float ZLC = *((float*)mxGetData(prhs[2]));
	const float dYL = *((float*)mxGetData(prhs[3]));
	const float dZL = *((float*)mxGetData(prhs[4]));
	const float SO = *((float*)mxGetData(prhs[5]));
    

	filtering(fpwd, Proj, YL, ZL, ViewN,
			PLC - 1.0f, ZLC - 1.0f, dYL, dZL, SO, 1);
}














